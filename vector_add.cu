#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>
#include <float.h>
#define FLOAT4(value) (reinterpret_cast<float4 *>(&(value))[0])
__global__ void vector_add(const float* A, const float* B, float* C, int N) {
    int idx = 4 * (blockIdx.x * blockDim.x + threadIdx.x);
    if (idx < N) {
        float4 reg_a = reinterpret_cast<float4*>(const_cast<float*>(&(A[idx])))[0];
        float4 reg_b = reinterpret_cast<float4*>(const_cast<float*>(&(B[idx])))[0];
        float4 reg_c;
        reg_c.x = reg_a.x + reg_b.x;
        reg_c.y = reg_a.y + reg_b.y;
        reg_c.z = reg_a.z + reg_b.z;
        reg_c.w = reg_a.w + reg_b.w;
        FLOAT4(C[idx]) = reg_c;
  }

}

// A, B, C are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* A, const float* B, float* C, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    vector_add<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, N);
    hipDeviceSynchronize();
}
